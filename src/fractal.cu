#include "hip/hip_runtime.h"
#include "raylib.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>


/* 
    Width of the screen
    Height of the screen, height also determines the amount of threads th use on the gpu
    Amount of blocks on the gpu to use
    Complexre is the real part of the complex number the Mandelbrot set is based on
    Complexim is the complex part of the complex number the Mandelbrot set is based on
 */
#define WIDTH  1900
#define HEIGHT 1024
#define AMOUNT 100
#define COMPLEXRE -0.4
#define COMPLEXIM 0.6


#if HEIGHT > 1024
    #error "Height is too big for the number of threads!"
#endif

// CUDA kernel to iterate through the Mandelbrot set on GPU
__global__ void iterategpu(double* imagine, double* real, unsigned char* C, int index){
    unsigned char i = 0;
    int thread = threadIdx.x;
    int block = blockIdx.x;
    double re = real[(index * AMOUNT + block) * HEIGHT + thread];
    double im = imagine[(index * AMOUNT + block) * HEIGHT + thread];

    // Iterate through the Mandelbrot set
    while (re * re + im * im < 2.0 && i < 200) {
        i++;
        double temp = re * re - im * im + COMPLEXRE;
        im = 2.0 * re * im + COMPLEXIM;
        re = temp;
    }

    // Store the iteration count in the output array
    C[(index * AMOUNT + block) * HEIGHT + thread] = i;
}

// Arrays for GPU computations
unsigned char screen[WIDTH * HEIGHT];
double inputIm[WIDTH * HEIGHT];
double inputRe[WIDTH * HEIGHT];


// Function to render the Mandelbrot set on GPU
void render(int offsetX, int offsetY, double zoom, double* inRe, double* inIm, unsigned char* out) {
    // Generate Mandelbrot set coordinates
    for (int i = 0; i < WIDTH; i++) {
        for (int j = 0; j < HEIGHT; j++) {
            double re = (double)(offsetX + i - (WIDTH / 2)) * zoom;
            double im = (double)(offsetY + j - (HEIGHT / 2)) * zoom;

            inputRe[i * HEIGHT + j] = re;
            inputIm[i * HEIGHT + j] = im;
        }
    }

    // Copy Mandelbrot set coordinates to GPU
    hipMemcpy(inRe, inputRe, sizeof(inputRe), hipMemcpyHostToDevice);
    hipMemcpy(inIm, inputIm, sizeof(inputIm), hipMemcpyHostToDevice);

    // Launch GPU kernel for Mandelbrot set computation
    for (int i = 0; i < WIDTH / AMOUNT; i++) {
        iterategpu <<<AMOUNT, HEIGHT>>> (inRe, inIm, out, i);
        hipDeviceSynchronize();
    }

    // Copy the result back to the CPU
    hipMemcpy(screen, out, sizeof(screen), hipMemcpyDeviceToHost);
}

// Function to draw the Mandelbrot set using raylib
void draw(int x, int y, double zoom, double* inRe, double* inIm, unsigned char* out) {
    BeginDrawing();
    unsigned char colorvalue;

    // Render the Mandelbrot set on GPU and copy it to the CPU
    render(x, y, zoom, inRe, inIm, out);

    // Draw the fractal in raylib
    for (int i = 0; i < WIDTH; i++) {
        for (int j = 0; j < HEIGHT; j++) {
            colorvalue = screen[i * HEIGHT + j];
            Color color = (Color){colorvalue, 0, (unsigned char)(2*colorvalue), 255};
            DrawPixel(i, j, color);
        }
    }

    EndDrawing();
}

int main() {
    double zoom = 4 / (double)HEIGHT;
    int x = 0;
    int y = 0;

    // Pointers to the VRAM
    double* inRe = 0;
    double* inIm = 0;
    unsigned char* out = 0;

    // Allocate memory on GPU
    hipMalloc(&inIm, sizeof(inputIm));
    hipMalloc(&inRe, sizeof(inputRe));
    hipMalloc(&out, sizeof(screen));

    // Initialize raylib window
    InitWindow(WIDTH, HEIGHT, "Fractal");
    SetTargetFPS(20);
    draw(x, y, zoom, inRe, inIm, out);

    // Main loop
    while (!WindowShouldClose()) {
        // Handle mouse input to zoom and pan the Mandelbrot set
        if (IsMouseButtonPressed(MOUSE_BUTTON_LEFT)) {
            Vector2 mousePosition = GetMousePosition();
            zoom *= 0.9;
            mousePosition.x -= WIDTH / 2;
            mousePosition.y -= HEIGHT / 2;

            mousePosition.x *= 0.0004 / zoom;
            mousePosition.y *= 0.0004 / zoom;

            x += mousePosition.x;
            y += mousePosition.y;

            draw(x, y, zoom, inRe, inIm, out);
        }
        // left mouse button zooms out
        else if (IsMouseButtonPressed(MOUSE_BUTTON_RIGHT)) {
            zoom *= 1.1;
            draw(x, y, zoom, inRe, inIm, out);
        }

        BeginDrawing();
        // The following line is required for raylib
        EndDrawing();
    }

    // Close raylib window
    CloseWindow();

    // Free memory on GPU
    hipFree(inIm);
    hipFree(inRe);
    hipFree(out);

    return 0;
}
